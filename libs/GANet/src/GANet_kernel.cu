#include "hip/hip_runtime.h"
//#include <torch/torch.h>
//#include <torch/serialize/tensor.h>
//#include <ATen/ATen.h>
//#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>


#define CUDA_NUM_THREADS 256 
#define THREADS_PER_BLOCK 64 

#define DIM0(TENSOR) ((TENSOR).x)
#define DIM1(TENSOR) ((TENSOR).y)
#define DIM2(TENSOR) ((TENSOR).z)
#define DIM3(TENSOR) ((TENSOR).w)

#define DIM3_INDEX(TENSOR, xx, yy, zz, ww) ((TENSOR)[((xx) * (TENSOR##_stride.x)) + ((yy) * (TENSOR##_stride.y)) + ((zz) * (TENSOR##_stride.z)) + ((ww) * (TENSOR##_stride.w))])

#ifdef __cplusplus
    extern "C" {
#endif



__global__ void Max (const int n, const float *top_temp, float *top_data, float *mask,
     const int mask_index){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  if (top_data[index] < top_temp[index])
    {
      top_data[index] = top_temp[index];
      mask[index] = mask_index;
    }
}

__global__ void get_temp_grad (const int n, const float *gradOutput, const float *mask,
	       float *top_grad, const int mask_index){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  if (((int) mask[index]) == mask_index)
    top_grad[index] = gradOutput[index];
}

__global__ void MaxDepth (const int n, const float *bottom_data, const int step,
	  const int depth, float *idx){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int base = index / step * step * depth + index % step;
  int k = 0;
  for (int i = 1; i < depth; i++)
    if (bottom_data[base + k * step] < bottom_data[base + i * step])
      k = i;
  idx[index] = k;
}

__global__ void sga_down_forward (const int n, const float *filters, const int height,
		  const int width, const int depth, const int wsize,
		  float *top_data){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
//  int wsize=radius+1;
  int base = index / width * step * depth + index % width;	//up->down
  int fbase = index / width * step * wsize + index % width;

  int kp = 0;

  for (int row = 0; row < height; row++)
    {
      int shift = fbase + row * width;

      int base0 = base + row * width;
      int k = kp;
      kp = 0;

/*        if(row-1>=0)
            for(int i = 1; i < depth; i++){
	        if(top_data[base0-width+k*step]<top_data[base0-width+i*step])
		    k = i;
*/
      for (int d = 0; d < depth; d++)
	{
	  float temp = 0;
	  int location = base0 + d * step;
	  temp += top_data[location] * filters[shift];
	  if (row - 1 >= 0)
	    temp += top_data[location - width] * filters[shift + step];
	  else
	    temp += top_data[location] * filters[shift + step];

	  if (row - 1 >= 0 && d - 1 >= 0)
	    temp +=
	      top_data[location - width - step] * filters[shift + 2 * step];
	  else
	    temp += top_data[location] * filters[shift + 2 * step];
	  if (row - 1 >= 0 && d + 1 < depth)
	    temp +=
	      top_data[location - width + step] * filters[shift + 3 * step];
	  else
	    temp += top_data[location] * filters[shift + 3 * step];
	  if (row - 1 >= 0)
	    temp +=
	      top_data[base0 - width + k * step] * filters[shift + 4 * step];
	  else
	    temp += top_data[location] * filters[shift + 4 * step];
	  top_data[location] = temp;

	  if (top_data[base0 + kp * step] < temp)
	    kp = d;

	}
    }
}

__global__ void sga_down_data_backward (const int n, const float *filters, float *top_diff,
			const float *idx, const int height, const int width,
			const int depth, const int wsize, float *bottom_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / width * step * depth + index % width;	//up->down
  int fbase = index / width * step * wsize + index % width;
//1
  int base_idx = index / width * step + index % width;
//
  for (int row = height - 1; row >= 0; row--)
    {
      int shift = fbase + row * width;
      for (int d = 0; d < depth; d++)
	{
	  int location = base + d * step + row * width;
	  float temp = top_diff[location];
	  if (row + 1 < height)
	    temp +=
	      top_diff[location + width] * filters[shift + width + step];

	  if (row + 1 < height && d + 1 < depth)
	    temp +=
	      top_diff[location + width + step] * filters[shift + width +
							  2 * step];
	  if (row + 1 < height && d - 1 >= 0)
	    temp +=
	      top_diff[location + width - step] * filters[shift + width +
							  3 * step];
	  top_diff[location] = temp;
	  bottom_diff[location] += temp * filters[shift];
	}
//2
      if (row + 1 < height)
	{
	  int k = idx[base_idx + row * width];
	  int location = base + k * step + row * width;
	  float temp = 0;
	  for (int d = 0; d < depth; d++)
	    temp +=
	      top_diff[base + row * width + width +
		       d * step] * filters[shift + width + 4 * step];
	  top_diff[location] += temp;
	  bottom_diff[location] += temp * filters[shift];
	}
//2

    }

/*	for(int d = 0; d < depth; d ++){
		int shift = fbase;
		int location = base + d * step;
		bottom_diff[location] += top_diff[location] * (filters[shift + step] + filters[shift + 2*step] + filters[shift + 3*step] + filters[shift + 4*step]); 
 //       bottom_diff[location] += top_diff[location];
		shift += width;
		location += width;
		bottom_diff[location] += top_diff[location] * filters[shift + 2*step];	
	}
	for(int row=1;row<height;row++){
		int location = base + row * width;
		int shift = fbase + row * width;
		bottom_diff[location] += top_diff[location] * filters[shift + 3*step]; 
		location += (depth - 1)*step;
		bottom_diff[location] += top_diff[location] * filters[shift + 4*step]; 
	}
*/
  for (int row = 0; row < height; row++)
    {
      int location = base + row * width;
      int shift = fbase + row * width;
      bottom_diff[location] += top_diff[location] * filters[shift + 2 * step];
      location += (depth - 1) * step;
      bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
    }
}

__global__ void sga_down_weight_backward (const int n, const float *bottom_data,
			  const float *top_data, const float *temp_diff,
			  const float *idx, const int height, const int width,
			  const int depth, const int wsize,
			  float *filters_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / step * step * depth + index % step;	//up->down
  int fbase = index / step * step * wsize + index % step;

  int row = index % step / width;
  for (int i = 0; i < depth; i++)
    filters_diff[fbase] +=
      temp_diff[base + i * step] * bottom_data[base + i * step];
  if (row - 1 >= 0)
    {
      int location = fbase + step;
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + i * step - width];

      location = fbase + 2 * step;
      filters_diff[location] += temp_diff[base] * bottom_data[base];
      for (int i = 1; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i - 1) * step -
						width];

      location = fbase + 3 * step;
      filters_diff[location] +=
	temp_diff[base + (depth - 1) * step] * bottom_data[base +
							   (depth -
							    1) * step];
      for (int i = 0; i < depth - 1; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i + 1) * step -
						width];
    }
/*
    else{
		for(int i=0; i<depth; i++){
			float temp = temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		    filters_diff[fbase + 3*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + 4*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		}

	}
*/
//1
  if (row - 1 >= 0)
    {
      int location = fbase + 4 * step;
      int k = idx[index - width];
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + k * step - width];
    }
//
/*
    else{
		int location = fbase + 2*step;
		for(int i=0; i<depth; i++)
			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}
*/
}



__global__ void sga_up_forward (const int n, const float *filters, const int height,
		const int width, const int depth, const int wsize,
		float *top_data){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= n)
    {
      return;
    }
  int step = height * width;
  //   int wsize=radius+1;

  int base = index / width * step * depth + index % width;	//up->down
  int fbase = index / width * step * wsize + index % width;

  int kp = 0;			//1

  for (int row = height - 1; row >= 0; row--)
    {
      int shift = fbase + row * width;
//2
      int base0 = base + row * width;
      int k = kp;
      kp = 0;
//2
      for (int d = 0; d < depth; d++)
	{
	  float temp = 0;
	  int location = base + d * step + row * width;
	  temp += top_data[location] * filters[shift];
	  if (row + 1 < height)
	    temp += top_data[location + width] * filters[shift + step];
	  else
	    temp += top_data[location] * filters[shift + step];

	  if (row + 1 < height && d - 1 >= 0)
	    temp +=
	      top_data[location + width - step] * filters[shift + 2 * step];
	  else
	    temp += top_data[location] * filters[shift + 2 * step];
	  if (row + 1 < height && d + 1 < depth)
	    temp +=
	      top_data[location + width + step] * filters[shift + 3 * step];
	  else
	    temp += top_data[location] * filters[shift + 3 * step];

//3
	  if (row + 1 < height)
	    temp +=
	      top_data[base0 + width + k * step] * filters[shift + 4 * step];
	  else
	    temp += top_data[location] * filters[shift + 4 * step];
	  top_data[location] = temp;

	  if (top_data[base0 + kp * step] < temp)
	    kp = d;
//3

	}
    }
}

__global__ void sga_up_data_backward (const int n, const float *filters, float *top_diff,
		      const float *idx, const int height, const int width,
		      const int depth, const int wsize, float *bottom_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / width * step * depth + index % width;	//up->down
  int fbase = index / width * step * wsize + index % width;

//1
  int base_idx = index / width * step + index % width;
//
  for (int row = 0; row < height; row++)
    {
      int shift = fbase + row * width;
      for (int d = 0; d < depth; d++)
	{
	  int location = base + d * step + row * width;
	  float temp = top_diff[location];
	  if (row - 1 >= 0)
	    temp +=
	      top_diff[location - width] * filters[shift - width + step];
	  if (row - 1 >= 0 && d + 1 < depth)
	    temp +=
	      top_diff[location - width + step] * filters[shift - width +
							  2 * step];
	  if (row - 1 >= 0 && d - 1 >= 0)
	    temp +=
	      top_diff[location - width - step] * filters[shift - width +
							  3 * step];
	  top_diff[location] = temp;
	  bottom_diff[location] += temp * filters[shift];
	}

//2
      if (row - 1 >= 0)
	{
	  int k = idx[base_idx + row * width];
	  int location = base + k * step + row * width;
	  float temp = 0;
	  for (int d = 0; d < depth; d++)
	    temp +=
	      top_diff[base + row * width - width +
		       d * step] * filters[shift - width + 4 * step];
	  top_diff[location] += temp;
	  bottom_diff[location] += temp * filters[shift];
	}
//2
    }

/*	for(int d = 0; d < depth; d ++){
		int shift = fbase + width*(height-1);
		int location = base + width*(height-1) + d * step;
		bottom_diff[location] += top_diff[location] * (filters[shift + step] + filters[shift + 2*step] + filters[shift + 3*step] + filters[shift + 4*step]); 
//        bottom_diff[location] += top_diff[location];
		shift -= width;
		location -= width;
		bottom_diff[location] += top_diff[location] * filters[shift + 2*step];	
	}
	for(int row=0;row<height-1;row++){
		int shift = fbase + row * width;
		int location = base + row * width;
		bottom_diff[location] += top_diff[location] * filters[shift + 3*step]; 
		location += (depth - 1)*step;
		bottom_diff[location] += top_diff[location] * filters[shift + 4*step]; 
	}*/
  for (int row = 0; row < height; row++)
    {
      int shift = fbase + row * width;
      int location = base + row * width;
      bottom_diff[location] += top_diff[location] * filters[shift + 2 * step];
      location += (depth - 1) * step;
      bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
    }
}

__global__ void sga_up_weight_backward (const int n, const float *bottom_data,
			const float *top_data, const float *temp_diff,
			const float *idx, const int height, const int width,
			const int depth, const int wsize, float *filters_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / step * step * depth + index % step;	//up->down
  int fbase = index / step * step * wsize + index % step;

  int row = index % step / width;
  for (int i = 0; i < depth; i++)
    filters_diff[fbase] +=
      temp_diff[base + i * step] * bottom_data[base + i * step];
  if (row + 1 < height)
    {
      int location = fbase + step;
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + i * step + width];

      location = fbase + 2 * step;
      filters_diff[location] += temp_diff[base] * bottom_data[base];
      for (int i = 1; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i - 1) * step +
						width];

      location = fbase + 3 * step;
      filters_diff[location] +=
	temp_diff[base + (depth - 1) * step] * bottom_data[base +
							   (depth -
							    1) * step];
      for (int i = 0; i < depth - 1; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i + 1) * step +
						width];
    }
/*
    else{
		//int location = fbase + step;
		for(int i=0; i<depth; i++){
			float temp = temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		    filters_diff[fbase + 3*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + 4*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		}
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];		
//		location = fbase + 3*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
//		
//		location = fbase + 4*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}*/
//1
  if (row + 1 < height)
    {
      int location = fbase + 4 * step;
      int k = idx[index + width];
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + k * step + width];
    }
//

/*
    else{
		int location = fbase + 2*step;
		for(int i=0; i<depth; i++)
			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}*/
}

__global__ void sga_right_forward (const int n, const float *filters, const int height,
		   const int width, const int depth, const int wsize,
		   float *top_data){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  // 注意,由于n = N * C * H，从这个if可以看出，作者共使用n = N * C * H个线程来处理一个batch
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  //   int wsize=radius+1;
  // 待聚合的代价体（top_data）维度为[N, C, D, H, W] = [N, 32, Dmax(=192/3), H, W],
  // 注意：n = N * C * H = N*32*H,从作者的意图来看，使用一个线程处理"一个样本的、一行像素的、所有视差的"
  // （即D维度和W维度，W维度体现了从左到右的聚合方向）的代价聚合
  // base用于top_data（原始聚合代价）的索引
  int base = index / height * step * depth + (index % height) * width;	//这是top_data[N, C, D=0, H, W=0]的内存索引，从此代价开始（遍历W和D）聚合。
  // 聚合权重（filters）的维度为[N, C, wsize(=5), H, W] = (N, 32, 5, H, W), wsize维度对应论文公式（5）的权重w0~w4.
  int fbase = index / height * step * wsize + (index % height) * width; //filters[N, C, wsize=0, H, W=0]的内存索引，从此权重开始（遍历wsize和W）聚合。

  int kp = 0;
  // 遍历W维度，即对当前行像素的匹配代价，从左到右进行聚合
  for (int col = 0; col < width; col++)
    {
      int shift = fbase + col;
//2
      int base0 = base + col;
      // k用于保存当前像素的前一个像素的最大聚合代价对应的视差
      int k = kp;
      kp = 0;
//2
     // 遍历D(视差)维度，即
      for (int d = 0; d < depth; d++)
	{
	  float temp = 0;
	  int location = base + d * step + col;
	  // 论文公式（5）的第一项
	  temp += top_data[location] * filters[shift];
	  // 论文公式（5）的第二项
	  if (col - 1 >= 0)
	    temp += top_data[location - 1] * filters[shift + step];
	  else
	    temp += top_data[location] * filters[shift + step];
      // 论文公式（5）的第三项
	  if (col - 1 >= 0 && d - 1 >= 0)  // 注意，由于公式（5）的要用到d-1，这里d=0时需特殊处理, 特别注意反向传播时的处理，要与这里对应！！！
	    temp += top_data[location - 1 - step] * filters[shift + 2 * step];
	  else
	    temp += top_data[location] * filters[shift + 2 * step];
	  // 论文公式（5）的第四项
	  if (col - 1 >= 0 && d + 1 < depth) // 注意，由于公式（5）的要用到d+1，这里d=depth-1时需特殊处理, 特别注意反向传播时的处理，要与这里对应！！！
	    temp += top_data[location - 1 + step] * filters[shift + 3 * step];
	  else
	    temp += top_data[location] * filters[shift + 3 * step];

//3
      // 论文公式（5）的第五项。 变量k保存着当前像素的前一个像素的最大聚合代价对应的视差（为公式（5）中的max函数服务）
	  if (col - 1 >= 0)
	    temp +=
	      top_data[base0 - 1 + k * step] * filters[shift + 4 * step];
	  else
	    temp += top_data[location] * filters[shift + 4 * step];

	  top_data[location] = temp;
      // 记录当前像素的最大聚合代价对应的视差，供下一个像素使用。
	  if (top_data[base0 + kp * step] < temp)
	    kp = d;
//3
	}
    }
}

// SGA的反向传播核心代码：针对本层的输入数据（即待聚合的代价体）
// input.size()=[num,channel,depth,height,width]
//  int num = input.size(0);
//  int channel = input.size(1);
//  int depth = input.size(2);
//  int height = input.size(3);
//  int width = input.size(4);
//  int wsize = guidance_down.size(2);
//  n = num * channel * height;
// filters：聚合权重（聚合聚合方向为:左-->右）。已知量！！！
// idx:维度和input的空间维度相同（不包含depth即视差维度）[num,channel,height,width]，用于记录各个通道上各个像素的最大代价对应的视差。已知量。
// top_diff：loss对于本层输出量的导数（注意，已经考虑了公式（6），将未被选中为输出聚合代价的地方mask为0了（使用的是get_temp_grad（）函数））。已知量。
// bottom_diff：loss对于本层输入量input的导数，维度和input一致。待求量！！！
// g0/g1/g2/g3为四个方向的聚合权重矩阵，其维度为(N, 32, 5, H, W), 故wsize=5（对应公式（5）中的w0~w4）
// 故wsize=5（对应公式（5）中的w0~w5）
__global__ void sga_right_data_backward (const int n, const float *filters, float *top_diff,
			 const float *idx, const int height, const int width,
			 const int depth, const int wsize, float *bottom_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  // 由于n = num * channel * height以及下面的这个n和线程索引的比较，可以得出，作者用一个GPU线程处理一行像素的一个通道的代价聚合。
  if (index >= n)
    {
      return;
    }
  // 注意，由于input.size()=[num,channel,depth,height,width]，
  // 特定的一行像素的特定的通道对应的视差在input中的索引为[num, channel, :, height, :]，
  // 特定的一个像素的特定的通道对应的视差在input中的索引为[num, channel, :, height, width]。
  // 同一个像素的特定通道的两个相邻视差的内存地址的差异。
  int step = height * width;
  // input的当前通道当前行的第一个像素的第一个视差值的内存索引。input的维度为[num,channel,depth,height,width]
  int base = index / height * step * depth + (index % height) * width;	//left->right
  // input的当前通道当前行的第一个像素的第一个聚合权重（一个像素5个权重，即w0~w4）的内存索引。聚合权重的维度为(N, 32, 5, H, W)
  int fbase = index / height * step * wsize + (index % height) * width;
//1
  // input的当前通道当前行的第一个像素的最大代价对应的视差值。idx的维度为[num,channel,height,width]
  int base_idx = index / height * step + (index % height) * width;
// 对当前通道当前行的梯度进行反向传播：从行尾向行首遍历，依次计算关于输入数据input的导数。[num, channel, :, height, ：]
  for (int col = width - 1; col >= 0; col--)
    {
      int shift = fbase + col; // 当前通道当前行第col个像素的第一个聚合权重的索引：[num, 32, 0, height, col]
      // 对于当前像素，遍历它的视差：[num, channel, :, height, width]
      for (int d = 0; d < depth; d++)
	{
	  int location = base + d * step + col; // 当前通道当前行第col个像素像素当前视差代价的索引：[num, channel, d, height, col]
	  float temp = top_diff[location]; // 公式12中的第一项
	  if (col + 1 < width)
	    temp += top_diff[location + 1] * filters[shift + 1 + step];  // 公式12中sum中的第一项
	  if (col + 1 < width && d + 1 < depth)
	    temp +=
	      top_diff[location + 1 + step] * filters[shift + 1 + 2 * step]; // 公式12中sum中的第二项
	  if (col + 1 < width && d - 1 >= 0)
	    temp +=
	      top_diff[location + 1 - step] * filters[shift + 1 + 3 * step]; // 公式12中sum中的第三项
	  top_diff[location] = temp;
	  bottom_diff[location] += (temp * filters[shift]); // 公式10。注意针对聚合方向r的累加。
	}
//2
      if (col + 1 < width)
	{
	  int k = idx[base_idx + col];  // 当前像素的最大代价对应的视差值索引。idx的维度为[num,channel,height,width]
	  int location = base + k * step + col; // 当前像素的最大代价值的索引
	  float temp = 0;
	  for (int d = 0; d < depth; d++)
	    temp +=
	      top_diff[base + col + 1 + d * step] * filters[shift + 1 +
							    4 * step]; // 公式13中sum中的第四项
	  top_diff[location] += temp;
	  bottom_diff[location] += temp * filters[shift]; // 公式10中
	}
//2     
    }
/*
	for(int d = 0; d < depth; d ++){
		int shift = fbase;// + width*(height-1);
		int location = base;// + width*(height-1) + d * step;
		bottom_diff[location] += top_diff[location] * (filters[shift + step] + filters[shift + 2*step] + filters[shift + 3*step] + filters[shift + 4*step]);
 //       bottom_diff[location] += top_diff[location];
		shift += 1;
		location += 1;
		bottom_diff[location] += top_diff[location] * filters[shift + 2*step];	
	}
	for(int col=1;col<width;col++){
		int shift = fbase + col;
		int location = base + col;
		bottom_diff[location] += top_diff[location] * filters[shift + 3*step]; 
		location += (depth - 1)*step;
		bottom_diff[location] += top_diff[location] * filters[shift + 4*step]; 
	}*/
  for (int col = 0; col < width; col++)
    {
      // 遍历当前通道的当前行的像素，处理边界部分的求导（视差为0时的导数，以及最大视差（d=depth-1）时的导数。这与公式（5）中使用了r-1, d-1有关。）
      // 在前项传播时亦做了特殊处理，这里与之对应。
      int shift = fbase + col;
      int location = base + col;
      bottom_diff[location] += top_diff[location] * filters[shift + 2 * step];  // 因为视差为0，即d=0时的导数尚未加入。
      location += (depth - 1) * step; // 当前像素的最大视差（depth-1）对应的代价的索引
      bottom_diff[location] += top_diff[location] * filters[shift + 3 * step]; // 因为视差为最大值，即d=Dmax时的导数尚未加入。
    }
}


// SGA的反向传播核心代码：针对本层的聚合权重求导
// input.size()=[num,channel,depth,height,width]
//  n = num * channel * height * width;
// bottom_data: 输入数据input。input.size()=[num,channel,depth,height,width]
// top_data：输出数据。维度和input一致[num,channel,depth,height,width]
// filters：聚合权重（聚合聚合方向为:左-->右）。已知量！！！
// idx:维度和input的空间维度相同（不包含depth即视差维度）[num,channel,height,width]，用于记录各个通道上各个像素的最大代价对应的视差。已知量。
// temp_diff：loss对于本层输出量的导数(注意，已经考虑了公式(6)，将未被选中为输出聚合代价的地方mask为0了(使用的是get_temp_grad()函数))。已知量。
// filters_diff：loss对于本层聚合权重的导数（左-->右聚合），维度和聚合权重一致。待求量！！！
// g0/g1/g2/g3为四个方向的聚合权重矩阵，其维度为(N, 32, 5, H, W), 故wsize=5（对应公式（5）中的w0~w4）
// 故wsize=5（对应公式（5）中的w0~w5）
__global__ void sga_right_weight_backward (const int n, const float *bottom_data,
			   const float *top_data, const float *temp_diff,
			   const float *idx, const int height,
			   const int width, const int depth, const int wsize,
			   float *filters_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  // 因为n = num * channel * height * width，由此可以看出，
  // 作者使用一个GPU线程处理一个像素一个通道的权重求导(求不同视差下的导数并求和，公式(11))：[num,channel, :, height,width]
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  // 当前像素，视差为0对应的代价索引，即[num,channel,0, h,w]
  int base = index / step * step * depth + index % step;	//left->right
  // 当前像素，视差为0对应的代价的聚合权重w0，即[num,channel,0, h,w]
  int fbase = index / step * step * wsize + index % step;

  //   int row = index%step/width;
  int col = index % step % width;
  // 遍历视差，求导数，求和。即公式(11)的第一个等式。
  for (int i = 0; i < depth; i++)
    filters_diff[fbase] +=
      temp_diff[base + i * step] * bottom_data[base + i * step];
  if (col - 1 >= 0)
    {
      // 遍历视差，求导数，求和。即公式(11)的第二个等式。
      int location = fbase + step;
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + i * step - 1];

      // 遍历视差，求导数，求和。即公式(11)的第三个等式。注意，这里针对d=0（=0）的情形，需要单独处理下。要和前向传播对应
      location = fbase + 2 * step;
      filters_diff[location] += temp_diff[base] * bottom_data[base];
      for (int i = 1; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i - 1) * step - 1];

      // 遍历视差，求导数，求和。即公式(11)的第四个等式。注意这里针对d=depth-1（i=depth-1）的情形需要单独处理下。要和前向传播对应
      location = fbase + 3 * step;
      filters_diff[location] +=
	temp_diff[base + (depth - 1) * step] * bottom_data[base +
							   (depth -
							    1) * step];
      for (int i = 0; i < depth - 1; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i + 1) * step - 1];
    }
/*
    else{
		//int location = fbase + step;
		for(int i=0; i<depth; i++){
			float temp = temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		    filters_diff[fbase + 3*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + 4*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		}
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];		
//		location = fbase + 3*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
//		
//		location = fbase + 4*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}*/
//1
  if (col - 1 >= 0)
    {
      // 公式(11)的第五项
      int location = fbase + 4 * step;
      int k = idx[index - 1];
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + k * step - 1];
    }
//
/*
    else{
		int location = fbase + 2*step;
		for(int i=0; i<depth; i++)
			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}*/
}

__global__ void sga_left_forward (const int n, const float *filters, const int height,
		  const int width, const int depth, const int wsize,
		  float *top_data){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= n)
    {
      return;
    }
  int step = height * width;
  //   int wsize=radius+1;

  int base = index / height * step * depth + (index % height) * width;	//up->down
  int fbase = index / height * step * wsize + (index % height) * width;

  int kp = 0;

  for (int col = width - 1; col >= 0; col--)
    {
      int shift = fbase + col;
//2
      int base0 = base + col;
      int k = kp;
      kp = 0;
//2
      for (int d = 0; d < depth; d++)
	{
	  float temp = 0;
	  int location = base + d * step + col;
	  temp += top_data[location] * filters[shift];
	  if (col + 1 < width)
	    temp += top_data[location + 1] * filters[shift + step];
	  else
	    temp += top_data[location] * filters[shift + step];

	  if (col + 1 < width && d - 1 >= 0)
	    temp += top_data[location + 1 - step] * filters[shift + 2 * step];
	  else
	    temp += top_data[location] * filters[shift + 2 * step];
	  if (col + 1 < width && d + 1 < depth)
	    temp += top_data[location + 1 + step] * filters[shift + 3 * step];
	  else
	    temp += top_data[location] * filters[shift + 3 * step];

//3
	  if (col + 1 < width)
	    temp +=
	      top_data[base0 + 1 + k * step] * filters[shift + 4 * step];
	  else
	    temp += top_data[location] * filters[shift + 4 * step];
	  top_data[location] = temp;

	  if (top_data[base0 + kp * step] < temp)
	    kp = d;
//3
	}
    }
}

__global__ void sga_left_data_backward (const int n, const float *filters, float *top_diff,
			const float *idx, const int height, const int width,
			const int depth, const int wsize, float *bottom_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / height * step * depth + (index % height) * width;	//up->down
  int fbase = index / height * step * wsize + (index % height) * width;
//1
  int base_idx = index / height * step + (index % height) * width;
//
  for (int col = 0; col < width; col++)
    {
      int shift = fbase + col;
      for (int d = 0; d < depth; d++)
	{
	  int location = base + d * step + col;
	  float temp = top_diff[location];
	  if (col - 1 >= 0)
	    temp += top_diff[location - 1] * filters[shift - 1 + step];
	  if (col - 1 >= 0 && d + 1 < depth)
	    temp +=
	      top_diff[location - 1 + step] * filters[shift - 1 + 2 * step];
	  if (col - 1 >= 0 && d - 1 >= 0)
	    temp +=
	      top_diff[location - 1 - step] * filters[shift - 1 + 3 * step];
	  top_diff[location] = temp;
	  bottom_diff[location] += temp * filters[shift];
	}
//2
      if (col - 1 >= 0)
	{
	  int k = idx[base_idx + col];
	  int location = base + k * step + col;
	  float temp = 0;
	  for (int d = 0; d < depth; d++)
	    temp +=
	      top_diff[base + col - 1 + d * step] * filters[shift - 1 +
							    4 * step];
	  top_diff[location] += temp;
//top_diff[base + col - 1 + d*step] * filters[shift - 1 + 4*step];
	  bottom_diff[location] += temp * filters[shift];
	}
//2             
    }
/*
	for(int d = 0; d < depth; d ++){
		int shift = fbase + width-1;// + width*(height-1);
		int location = base + width-1;// + width*(height-1) + d * step;
		bottom_diff[location] += top_diff[location] * (filters[shift + step] + filters[shift + 2*step] + filters[shift + 3*step] + filters[shift + 4*step]); 
//        bottom_diff[location] += top_diff[location];
		shift -= 1;
		location -= 1;
		bottom_diff[location] += top_diff[location] * filters[shift + 2*step];	
	}
	for(int col=0;col<width-1;col++){
		int shift = fbase + col;
		int location = base + col;
		bottom_diff[location] += top_diff[location] * filters[shift + 3*step]; 
		location += (depth - 1)*step;
		bottom_diff[location] += top_diff[location] * filters[shift + 4*step]; 
	}*/
  for (int col = 0; col < width; col++)
    {
      int shift = fbase + col;
      int location = base + col;
      bottom_diff[location] += top_diff[location] * filters[shift + 2 * step];
      location += (depth - 1) * step;
      bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
    }
}

__global__ void sga_left_weight_backward (const int n, const float *bottom_data,
			  const float *top_data, const float *temp_diff,
			  const float *idx, const int height, const int width,
			  const int depth, const int wsize,
			  float *filters_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int base = index / step * step * depth + index % step;	//up->down
  int fbase = index / step * step * wsize + index % step;

  //   int row = index%step/width;
  int col = index % step % width;
  for (int i = 0; i < depth; i++)
    filters_diff[fbase] +=
      temp_diff[base + i * step] * bottom_data[base + i * step];
  if (col + 1 < width)
    {
      int location = fbase + step;
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + i * step + 1];

      location = fbase + 2 * step;
      filters_diff[location] += temp_diff[base] * bottom_data[base];
      for (int i = 1; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i - 1) * step + 1];

      location = fbase + 3 * step;
      filters_diff[location] +=
	temp_diff[base + (depth - 1) * step] * bottom_data[base +
							   (depth -
							    1) * step];
      for (int i = 0; i < depth - 1; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + (i + 1) * step + 1];
    }
/*
    else{
		//int location = fbase + step;
		for(int i=0; i<depth; i++){
			float temp = temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		    filters_diff[fbase + 3*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
			filters_diff[fbase + 4*step] += temp; //temp_diff[base+i*step]*bottom_data[base+i*step];
		}
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];		
//		location = fbase + 3*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
//		
//		location = fbase + 4*step;
//		for(int i=0; i<depth; i++)
//			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}*/
//1
  if (col + 1 < width)
    {
      int location = fbase + 4 * step;
      int k = idx[index + 1];
      for (int i = 0; i < depth; i++)
	filters_diff[location] +=
	  temp_diff[base + i * step] * top_data[base + k * step + 1];
    }
//
/*
    else{
		int location = fbase + 2*step;
		for(int i=0; i<depth; i++)
			filters_diff[location] += temp_diff[base+i*step]*bottom_data[base+i*step];
	}
*/
}

void sga_kernel_forward (at::Tensor input, at::Tensor guidance_down,
		    at::Tensor guidance_up, at::Tensor guidance_right,
		    at::Tensor guidance_left, at::Tensor temp_out,
		    at::Tensor output, at::Tensor mask){

  int num = input.size(0);
  int channel = input.size(1);
  int depth = input.size(2);
  int height = input.size(3);
  int width = input.size(4);
  int wsize = guidance_down.size(2);

  //THCudaTensor_nElement(state, input);
  float *top_data = output.data<float>();
  float *top_temp = temp_out.data<float>();
  float *top_mask = mask.data<float>();

  const float *bottom_data = input.data<float>();
  const float *g0 = guidance_down.data<float>();
  const float *g1 = guidance_up.data<float>();
  const float *g2 = guidance_right.data<float>();
  const float *g3 = guidance_left.data<float>();

  int n = num * channel * width;
  int threads = (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
  int N = input.numel ();
//      hipStream_t stream = at::cuda::getCurrentCUDAStream(); 
  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_down_forward <<< threads, CUDA_NUM_THREADS >>> (n, g0, height, width,
						      depth, wsize, top_temp);
//      hipMemset( top_mask, 0, sizeof(float)*N);
  hipMemcpy (top_data, top_temp, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);

  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_up_forward <<< threads, CUDA_NUM_THREADS >>> (n, g1, height, width,
						    depth, wsize, top_temp);

  Max <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, top_data, top_mask, 1);

  n = num * channel * height;
  threads = (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;

  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_right_forward <<< threads, CUDA_NUM_THREADS >>> (n, g2, height, width,
						       depth, wsize,
						       top_temp);
  Max <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, top_data, top_mask, 2);

  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_left_forward <<< threads, CUDA_NUM_THREADS >>> (n, g3, height, width,
						      depth, wsize, top_temp);
  Max <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, top_data, top_mask, 3);

//      hipMemset( top_temp, 0, sizeof(float)*THCudaTensor_nElement(state, top_temp));

}

void sga_kernel_backward (at::Tensor input, at::Tensor guidance_down,
		     at::Tensor guidance_up, at::Tensor guidance_right,
		     at::Tensor guidance_left, at::Tensor temp_out,
		     at::Tensor mask, at::Tensor max_idx,
		     at::Tensor gradOutput, at::Tensor temp_grad,
		     at::Tensor gradInput, at::Tensor grad_down,
		     at::Tensor grad_up, at::Tensor grad_right,
		     at::Tensor grad_left){

  int num = input.size(0);
  int channel = input.size(1);
  int depth = input.size(2);
  int height = input.size(3);
  int width = input.size(4);
  int wsize = guidance_down.size(2);

  //THCudaTensor_nElement(state, input);
  float *top_grad = temp_grad.data<float>();
  float *top_temp = temp_out.data<float>();
  const float *top_mask = mask.data<float>();

  const float *bottom_data = input.data<float>();
  const float *grad_out = gradOutput.data<float>();

  const float *g0 = guidance_down.data<float>();
  const float *g1 = guidance_up.data<float>();
  const float *g2 = guidance_right.data<float>();
  const float *g3 = guidance_left.data<float>();

  float *grad0 = grad_down.data<float>();
  float *grad1 = grad_up.data<float>();
  float *grad2 = grad_right.data<float>();
  float *grad3 = grad_left.data<float>();
  float *grad_input = gradInput.data<float>();

  float *idx = max_idx.data<float>();

// input：本层的输入数据，即未经聚合的原始代价体。已知量。
// grad0：聚合权重的导数（聚合聚合方向为:下-->上）。待求量！！！
// grad1：聚合权重的导数（聚合聚合方向为:左-->右）。待求量！！！
// grad2：聚合权重的导数（聚合聚合方向为:上-->下）。待求量！！！
// grad3：聚合权重的导数（聚合聚合方向为:右-->左）。待求量！！！
// temp_grad：和input的维度相同，用于临时保存对input的导数！！！
// mask：维度和input相同。用于记录本层的输出数据（top_data）是从哪个聚合方向得到的（从四个方向选取最大聚合代价值，作为前向传播的输出）。已知量。
// max_idx:维度和input的空间维度相同（不包含depth即视差维度），用于记录各个通道上各个像素的最大代价对应的视差。已知量。
// gradInput：input的导数。维度和input一致。待求量！！！
// gradOutput：loss对于本层输出量的导数。已知量。
// g0/g1/g2/g3为四个方向的聚合权重矩阵，其维度为(N, 32, 5, H, W), 故wsize=5（对应公式（5）中的w0~w5）

  int N = input.numel ();
//      hipStream_t stream = at::cuda::getCurrentCUDAStream(); 

//backward for left             
  int n = num * channel * height;
//              hipMemcpy(top_temp, bottom_data, sizeof(float)*N, hipMemcpyDeviceToDevice);
//              sga_left_forward<<<(n + CUDA_NUM_THREADS - 1)/CUDA_NUM_THREADS, CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>
//              (n,g3,height,width,depth,wsize,top_temp);

  hipMemset (top_grad, 0, sizeof (float) * N);
  get_temp_grad <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, grad_out, top_mask, top_grad, 3);

  N = num * channel * width * height;
  MaxDepth <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, height * width, depth, idx);

  sga_left_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g3, top_grad, idx, height, width, depth, wsize,
			  grad_input);
  n = num * channel * width * height;
  sga_left_weight_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, bottom_data, top_temp, top_grad, idx, height,
			  width, depth, wsize, grad3);
//backward for down             
  N = input.numel ();
  n = num * channel * width;
  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_down_forward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g0, height, width, depth, wsize, top_temp);

  hipMemset (top_grad, 0, sizeof (float) * N);
  get_temp_grad <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, grad_out, top_mask, top_grad, 0);

  N = num * channel * width * height;
  MaxDepth <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, height * width, depth, idx);

  sga_down_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g0, top_grad, idx, height, width, depth, wsize,
			  grad_input);
  n = num * channel * width * height;
  sga_down_weight_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, bottom_data, top_temp, top_grad, idx, height,
			  width, depth, wsize, grad0);
// backward for up              
  N = input.numel ();
  n = num * channel * width;
  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  sga_up_forward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g1, height, width, depth, wsize, top_temp);

  hipMemset (top_grad, 0, sizeof (float) * N);
  get_temp_grad <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, grad_out, top_mask, top_grad, 1);
  N = num * channel * width * height;
  MaxDepth <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, height * width, depth, idx);

  sga_up_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g1, top_grad, idx, height, width, depth, wsize,
			  grad_input);
  n = num * channel * width * height;
  sga_up_weight_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, bottom_data, top_temp, top_grad, idx, height,
			  width, depth, wsize, grad1);
//backward for right            
  N = input.numel ();
  n = num * channel * height;
  hipMemcpy (top_temp, bottom_data, sizeof (float) * N,
	      hipMemcpyDeviceToDevice);
  // 从这里可以看出，作者使用n = num * channel * height个线程来做sga_right_forward。
  sga_right_forward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g2, height, width, depth, wsize, top_temp);

  hipMemset (top_grad, 0, sizeof (float) * N);
  get_temp_grad <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, grad_out, top_mask, top_grad, 2);

  N = num * channel * width * height;
  MaxDepth <<< (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (N, top_temp, height * width, depth, idx);

  sga_right_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, g2, top_grad, idx, height, width, depth, wsize,
			  grad_input);
  n = num * channel * width * height;
  sga_right_weight_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, bottom_data, top_temp, top_grad, idx, height,
			  width, depth, wsize, grad2);
}

__global__ void lga_filtering_forward (const int n, const float *bottom_data,
		       const float *filters, const int height,
		       const int width, const int channel, const int radius,
		       float *top_data){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
//    printf("OK\n");
//    printf("%d, %.2f, %.2f\n", index, bottom_data[index], top_data[index]);
  if (index >= n)
    {
      return;
    }
//    top_data[index]=1.0;
//    assert(0);
  int step = height * width;
  int wsize = 2 * radius + 1;
//      int fsize=wsize*wsize*3;
  int fbase =
    index / (step * channel) * (step * wsize * wsize * 3) + index % step;
  int row = index % step / width;
  int col = index % width;
  int depth = index / step % channel;
  for (int d = -1; d <= 1; d++)
    {
      for (int r = -radius; r <= radius; r++)
	{
	  for (int c = -radius; c <= radius; c++)
	    {
	      int rr = r + row;
	      int cc = c + col;
	      int dd = d + depth;
	      int shift = 0;
	      if (rr >= 0 && cc >= 0 && dd >= 0 && rr < height && cc < width
		  && dd < channel)
		shift = r * width + c + d * step;
	      int location =
		(d + 1) * (wsize * wsize) + (r + radius) * wsize + c + radius;
	      top_data[index] +=
		bottom_data[index + shift] * filters[fbase + location * step];
	    }
	}
    }
//        top_data[index]=1.0;
//        printf("%d, %d, %d, %.2f, %.2f\n", index, row, col, bottom_data[index], top_data[index]);
}

__global__ void lga_filter_backward (const int n, const float *bottom_data,
		     const float *top_diff, const int height, const int width,
		     const int channel, const int radius, float *filter_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int wsize = 2 * radius + 1;

  int base =
    index / (step * wsize * wsize * 3) * (step * channel) + index % step;
  int location = index / step % (wsize * wsize * 3);
  int d = location / (wsize * wsize) - 1;
  int r = (location / wsize) % wsize - radius;
  int c = location % wsize - radius;

  int rr = index % step / width + r;
  int cc = index % width + c;

  for (int i = 0; i < channel; i++)
    {
      int dd = i + d;
      if (rr >= 0 && cc >= 0 && dd >= 0 && rr < height && cc < width
	  && dd < channel)
	{
	  int shift = r * width + c + d * step;
	  filter_diff[index] +=
	    top_diff[base + i * step] * bottom_data[base + shift + i * step];
	}
      else
	filter_diff[index] +=
	  top_diff[base + i * step] * bottom_data[base + i * step];
    }


}

__global__ void lga_data_backward (const int n, const float *filters, const float *top_diff,
		   const int height, const int width, const int channel,
		   const int radius, float *bottom_diff){

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= n)
    {
      return;
    }
  int step = height * width;
  int wsize = 2 * radius + 1;
//      int fsize=wsize*wsize*3;
  int fbase =
    index / (step * channel) * (step * wsize * wsize * 3) + index % step;
  int row = index % step / width;
  int col = index % width;
  int depth = index / step % channel;
  for (int d = -1; d <= 1; d++)
    {
      for (int r = -radius; r <= radius; r++)
	{
	  for (int c = -radius; c <= radius; c++)
	    {
	      int rr = r + row;
	      int cc = c + col;
	      int dd = d + depth;
	      //      int shift = 0;
	      if (rr >= 0 && cc >= 0 && dd >= 0 && rr < height && cc < width
		  && dd < channel)
		{
		  int shift = r * width + c + d * step;
		  //      int fshift= r*width+c;
		  int location =
		    (-d + 1) * (wsize * wsize) + (-r + radius) * wsize - c +
		    radius;
		  bottom_diff[index] +=
		    top_diff[index + shift] * filters[fbase + r * width + c +
						      location * step];
		}
	      else
		{
		  int location =
		    (d + 1) * (wsize * wsize) + (r + radius) * wsize + c +
		    radius;
		  bottom_diff[index] +=
		    top_diff[index] * filters[fbase + location * step];
		}
	    }
	}
    }
}

void lga_forward (at::Tensor input, at::Tensor filters, at::Tensor output,
	     const int radius){

//        print_kernel<<<10, 10>>>();
//        hipDeviceSynchronize();
  //       int num=input->size(0);
  int channel = input.size(1);
  int height = input.size(2);
  int width = input.size(3);
  int n = input.numel ();
  //       printf("%d, %d, %d, %d, %d\n", height, width, channel, n, radius);
  //       hipStream_t stream = at::cuda::getCurrentCUDAStream();
/*        float *temp = new float[n];
        float *out = input.data<float>();
        hipMemcpy(temp,out,n*sizeof(float),hipMemcpyDeviceToHost);	
        for(int i=0;i<n;i++)
           printf("%.2f ", temp[i]);
*/
  lga_filtering_forward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, input.data<float>(), filters.data<float>(),
			  height, width, channel, radius,
			  output.data<float>());
  //     temp = new float[n];


}


void lga_backward (at::Tensor input, at::Tensor filters, at::Tensor gradOutput,
	      at::Tensor gradInput, at::Tensor gradFilters, const int radius){

//      int num=input->size(0);
  int channel = input.size(1);
  int height = input.size(2);
  int width = input.size(3);
//    hipStream_t stream = at::cuda::getCurrentCUDAStream();
  int n = filters.numel ();
  lga_filter_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, input.data<float>(),
			  gradOutput.data<float>(), height, width, channel,
			  radius, gradFilters.data<float>());
//    printf("%d, %d, %d, %d\n", height, width, channel, n);

  n = input.numel ();
  float *grad = gradInput.data<float>();
  hipMemset (grad, 0, sizeof (float) * n);
  lga_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, filters.data<float>(),
			  gradOutput.data<float>(), height, width, channel,
			  radius, grad);

}

void lga3d_forward (at::Tensor input, at::Tensor filters, at::Tensor output,
	       const int radius){

  //       int num=input->size(0);
  int channel = input.size(2);
  int height = input.size(3);
  int width = input.size(4);
  int n = input.numel ();
//        hipStream_t stream = at::cuda::getCurrentCUDAStream();
  lga_filtering_forward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, input.data<float>(), filters.data<float>(),
			  height, width, channel, radius,
			  output.data<float>());

}


void lga3d_backward (at::Tensor input, at::Tensor filters, at::Tensor gradOutput,
		at::Tensor gradInput, at::Tensor gradFilters,
		const int radius){

//      int num=input->size(0);
  int channel = input.size(2);
  int height = input.size(3);
  int width = input.size(4);
//    hipStream_t stream = at::cuda::getCurrentCUDAStream();
  int n = filters.numel ();
  lga_filter_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, input.data<float>(),
			  gradOutput.data<float>(), height, width, channel,
			  radius, gradFilters.data<float>());

  n = input.numel ();
  float *grad = gradInput.data<float>();
  hipMemset (grad, 0, sizeof (float) * n);
  lga_data_backward <<< (n + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS,
    CUDA_NUM_THREADS >>> (n, filters.data<float>(),
			  gradOutput.data<float>(), height, width, channel,
			  radius, grad);

}



#ifdef __cplusplus
    }
#endif
